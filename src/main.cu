/**
 * Author: Bucky Frost
 * File: main.cpp
 * Purpose: main function for checkers AI.
 */

#include <iostream>
#include <vector>
#include <chrono>
#include "neural_network.hpp"
#include "g_neural_network.hpp"
#include "minimax.hpp"
using std::cout;
using std::endl;

std::vector<int> standard_network = {32, 40, 10, 1};

void create_NN();
void timing();

int main(int argc, char* argv[])
{
	return 0;
}

void create_NN()
{
        G_Neural_Network * base_case_network = new G_Neural_Network(standard_network);

	std::ofstream ofs("test_network_save.txt");

	boost::archive::text_oarchive oa(ofs);

	oa << base_case_network;
	ofs.close();

}

void timing()
{
        int n=32*40*8; // size of network... total number of floats
        G_Neural_Network * whitey = new G_Neural_Network(standard_network);

        // Run gpu stuff on shared space
        int nBlocks = 1;    // GPU thread blocks to run
        int blockDim = n;   // threads per block, should be 256 for best performance
	for (int ii = 0; ii < 100; ++ii) {
        	auto start_gpu = std::chrono::high_resolution_clock::now();
		evaluate<<<nBlocks, blockDim>>>(whitey);    // evaluate on gpu
		hipDeviceSynchronize();    // wait to finish evaluation
        	auto end_gpu = std::chrono::high_resolution_clock::now();
        	cout << std::chrono::duration<double, std::nano> (end_gpu - start_gpu).count() << endl;
	}


}

/**
 * Author: Bucky Frost
 * File: main.cpp
 * Purpose: main function for checkers AI.
 */

#include <iostream>
#include <fstream>
using std::ifstream;
using std::ofstream;
#include <boost/archive/text_iarchive.hpp>
#include <boost/archive/text_oarchive.hpp>
#include <vector>
#include <chrono>
#include "neural_network.hpp"
#include "g_neural_network.hpp"
#include "minimax.hpp"
using std::cout;
using std::endl;

std::vector<int> standard_network = {32, 40, 10, 1};

void create_NN();
void timing();
void load_NN();

int main(int argc, char* argv[])
{
	create_NN();
	load_NN();

	return 0;
}

void load_NN()
{
	G_Neural_Network * test_case_network = new G_Neural_Network(standard_network);
	
	try {
		ifstream ifs("base_network_save.txt");
		int state = ifs.rdstate();
		cout << "FILE STATE: " << state << endl;
		boost::archive::text_iarchive ia(ifs);
		//ia >> test_case_network;
		ifs.close();
	} catch (const boost::archive::archive_exception& e) {
		cout << e.what();
	}
}

void create_NN()
{
        G_Neural_Network * base_case_network = new G_Neural_Network(standard_network);

	std::ofstream ofs("base_network_save.txt");

	boost::archive::text_oarchive oa(ofs);

	oa << base_case_network;
	ofs.close();

}

void timing()
{
        int n=32*40*8; // size of network... total number of floats
        G_Neural_Network * whitey = new G_Neural_Network(standard_network);

        // Run gpu stuff on shared space
        int nBlocks = 1;    // GPU thread blocks to run
        int blockDim = n;   // threads per block, should be 256 for best performance
	for (int ii = 0; ii < 100; ++ii) {
        	auto start_gpu = std::chrono::high_resolution_clock::now();
		evaluate<<<nBlocks, blockDim>>>(whitey);    // evaluate on gpu
		hipDeviceSynchronize();    // wait to finish evaluation
        	auto end_gpu = std::chrono::high_resolution_clock::now();
        	cout << std::chrono::duration<double, std::nano> (end_gpu - start_gpu).count() << endl;
	}


}

#include "hip/hip_runtime.h"
/**
 * File: neural_network.hpp
 * Purpose: Header file for the neural network.
 */

#include <vector>
using std::vector;
#include <fstream>
#include </opt/nvidia/cuda/include/hip/hip_runtime.h>
#include </opt/nvidia/cuda/include/hiprand.h>
#include "player.hpp"
#include "checker_board.hpp"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <random>
#include <boost/archive/text_oarchive.hpp>
#include <boost/archive/text_iarchive.hpp>

static std::random_device randomDevice;
static std::mt19937 random_value(randomDevice());
static std::uniform_real_distribution<> uniform_distribution(-1, 1);

struct network_node
{
	float _input;
	float _weight;

	__host__ __device__ network_node() : _input(1.0)
	{
	//	_weight = uniform_distribution(random_value);
	}

	__host__ __device__ void set_input(float val) { _input = val; }
	__host__ __device__ float node_value() { return _input*_weight; }

	__host__ __device__ bool operator==(const network_node & other) const
	{
		return (other._input == _input && other._weight == _weight);
	}
};
#include<hiprand/hiprand_kernel.h>


int n = 200;

__device__ float generate( hiprandState* globalState, int ind ) 
{
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}

__global__ void kernel(float* N, hiprandState* globalState, int n)
{
    // generate random numbers
    for(int i=0;i<40000;i++)
    {
        int k = generate(globalState, i) * 100000;
        while(k > n*n-1)
        {
            k-=(n*n-1);
        }
        N[i] = k;
    }
}

// ffrom stackoverlow
class Unified 
{
    public:
        // Allocate instances in CPU/GPU unified memory. Needs Kepler Architecture.
        void *operator new(size_t len)
        {
            void *ptr;
            hipMallocManaged(&ptr, len);
            return ptr;
        }
        void *operator new[](size_t size)
        {
            void *ptr;
            hipMallocManaged(&ptr, size);
            return ptr;
        }
        void operator delete(void *ptr) { hipFree(ptr); }
        void operator delete[](void *ptr) { hipFree(ptr); }

};

typedef vector<vector<network_node>> Network;
class G_Neural_Network : public Unified
{
public:

         __host__ __device__ G_Neural_Network(vector<int> network_specs) 
	{
		int network_layers = network_specs.size();

		for (int ii = 0; ii < network_layers; ++ii) 
                {
			_network.emplace_back(vector<network_node>(network_specs[ii]));
		}
	}


	// Feed forward the network to evaluate the checker board.
         __host__ __device__ void network_evaluate(float * weights)
	{
		
		int network_input_size = 32; 
		int input_layer = 0;

		thrust::device_vector<float> temp(network_input_size);

		thrust::fill(temp.begin(), temp.end(), 1.0);

                for (int ii = 0; ii < network_input_size; ++ii) 
                {
                    //_network[input_layer][ii].set_input(board_input[ii]);
                    _network[input_layer][ii].set_input(temp[ii]);
                }

                for (int layer = input_layer + 1; layer < _network.size(); ++layer) 
                {
                    for (int column = 0; column < _network[layer].size(); ++column) 
                    {
                        for (int ii = 0; ii < _network[layer - 1].size(); ++ii)
                            _network[layer][column]._input *=
                                    sigmoid(_network[layer - 1][ii].node_value());
                    }
                }

		auto network_output = _network.back();
		float evaluation_value = _network.back().back().node_value();
		//return sigmoid(evaluation_value);
	}
/*	
	__host__ __device__  bool operator==(const G_Neural_Network & other) const 
        { return other._network == _network; }
	__host__ __device__  bool operator!=(const G_Neural_Network & other) const 
        { return !(*this == other); }
*/

private:
	Network _network;

	__host__ __device__  G_Neural_Network() = default;
	__host__ __device__  float sigmoid(float input) { return input/(1.0 + abs(input)); }
#if 0
	friend class boost::serialization::access;
	template<class Archive>
	 void serialize(Archive & ar, const unsigned int version) 
        { ar & _network; }
#endif
};

/**
 * Author: Bucky Frost
 * File: main.cpp
 * Purpose: main function for checkers AI.
 */

#include <iostream>
#include <vector>
#include <chrono>
//#include "g_neural_network.hpp"
#include "minimax.hpp"
#include "checker_board.hpp"

using std::cout;
using std::endl;
 __device__ void network_evaluate(float * weights)
{
	


}

__global__ void evaluate(float * layer0, float * layer1, float * layer2, float * layer3, float * t0, float * t1,  float *t2)
{
	// use alls threads	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	//network_evaluate(weights); 
	*t0 = *t0 + layer0[idx];	
	*t1 = *t1 + layer1[idx];	
	*t2 = *t2 + layer2[idx];	
}

__global__ void final(float * layer3, float * t0, float * t1,  float *t2)
{
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	layer3[idx] = *t0 + *t1 + *t2 + layer3[idx];
}
// Precondition - It doesn't make sense to have less than 32 as first layer
// for checkers evaluations. It doesn't make sense to have more than 1 as
// final layer. Thus, those will automatically be set for any non-default
// arg parameters passed into main.
std::vector<int> argparse(int argc, char* argv[])
{
    std::vector<int> layer_sizes{32, 40, 10, 1};
    if (argc > 1)
    {
        layer_sizes.resize(argc + 1);
        layer_sizes.push_back(32);
        for (unsigned int count = 1; count < argc; ++count)
        {
            layer_sizes[count] = std::stoi(argv[count]);
        }
        layer_sizes.push_back(1);

    }
    return layer_sizes;
}


void create_NN(std::vector<int>);
void timing(std::vector<int>);

int main(int argc, char* argv[])
{
        std::vector<int> neural_network_layers = argparse(argc, argv);
	    int N=40000;

    hiprandState* devStates;
    hipMalloc ( &devStates, N*sizeof( hiprandState ) );

    // setup seeds
    setup_kernel <<< 1, N >>> ( devStates,unsigned(time(NULL)) );

    float N2[40000];
    float* N3;


	int n1 = 50000;
	int n2 = 12500;
	int n3 = 1250;
	float nr1[n1];
	float nr2[n2];
	float nr3[n3];
	
	float* nr1_val;
	float* nr2_val;
	float* nr3_val;
    hipMalloc((void**) &N3, sizeof(float)*N);
    hipMalloc((void**) &nr1_val, sizeof(float)*n1);
    hipMalloc((void**) &nr2_val, sizeof(float)*n2);
    hipMalloc((void**) &nr3_val, sizeof(float)*n3);

//get some random values into the first param
    kernel<<<1,1>>> (N3, devStates, n);
    kernel<<<1,1>>> (nr1_val, devStates, n);
    kernel<<<1,1>>> (nr2_val, devStates, n);
    kernel<<<1,1>>> (nr3_val, devStates, n);
float tt = 1.0;
float ttu = 1.0;
float tti = 1.0;
float * t0 = &tt;
float * t1 = &ttu;
float * t2 = &tti;

	evaluate<<<1,1>>> (N3, nr1_val, nr2_val, nr3_val, t0, t1, t2);
	final<<<1,1>>> (nr3_val, t0,t1,t2);

    hipMemcpy(nr1, nr1_val, sizeof(float)*n1, hipMemcpyDeviceToHost);
    hipMemcpy(nr2, nr2_val, sizeof(float)*n2, hipMemcpyDeviceToHost);
    hipMemcpy(nr3, nr3_val, sizeof(float)*n3, hipMemcpyDeviceToHost);
	for ( int ii = 0; ii < n1; ++ii)
		cout << nr1[0] << endl;
	for ( int ii = 0; ii < n1; ++ii)
		cout << nr2[0] << endl;
	for ( int ii = 0; ii < n3; ++ii)
		cout << nr3[0] << endl;

/*
    for(int i=0;i<N;i++)
    {
        cout<<N2[i]<<endl;
    }
*/

     //   timing(neural_network_layers);
	return 0;
}

#if 0
void create_NN(std::vector<int> layers)
{
        G_Neural_Network * base_case_network = new G_Neural_Network(layers);

	std::ofstream ofs("test_network_save.txt");

	boost::archive::text_oarchive oa(ofs);

	oa << base_case_network;
	ofs.close();

}

void timing(std::vector<int> layers)
{
        G_Neural_Network * player = new G_Neural_Network(layers);
        int n = 0;
        for (auto i : layers) { n += i; } // sum of NN nodes.

        // Run gpu stuff on shared space
        int nBlocks = 1;    // GPU thread blocks to run
        int blockDim = n;   // threads per block, should be 256 for best performance
	for (int ii = 0; ii < 100; ++ii) {
        	auto start_gpu = std::chrono::high_resolution_clock::now();
		evaluate<<<nBlocks, blockDim>>>(player);    // evaluate on gpu
		hipDeviceSynchronize();    // wait to finish evaluation
        	auto end_gpu = std::chrono::high_resolution_clock::now();
        	cout << std::chrono::duration<double, std::nano> (end_gpu - start_gpu).count() << endl;
	}


}
#endif

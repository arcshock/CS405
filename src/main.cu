/**
 * Author: Bucky Frost
 * File: main.cpp
 * Purpose: main function for checkers AI.
 */

#include <iostream>
#include <vector>
#include <chrono>
#include "neural_network.hpp"
#include "g_neural_network.hpp"
#include "minimax.hpp"
using std::cout;
using std::endl;

// Precondition - It doesn't make sense to have less than 32 as first layer
// for checkers evaluations. It doesn't make sense to have more than 1 as
// final layer. Thus, those will automatically be set for any non-default
// arg parameters passed into main.
std::vector<int> argparse(int argc, char* argv[])
{
    std::vector<int> layer_sizes{32, 40, 10, 1};
    if (argc > 1)
    {
        layer_sizes.resize(argc + 1);
        layer_sizes.push_back(32);
        for (unsigned int count = 1; count < argc; ++count)
        {
            layer_sizes[count] = std::stoi(argv[count]);
        }
        layer_sizes.push_back(1);

    }
    return layer_sizes;
}


void create_NN(std::vector<int>);
void timing(std::vector<int>);

int main(int argc, char* argv[])
{
        std::vector<int> neural_network_layers = argparse(argc, argv);
        timing(neural_network_layers);
	return 0;
}

void create_NN(std::vector<int> layers)
{
        G_Neural_Network * base_case_network = new G_Neural_Network(layers);

	std::ofstream ofs("test_network_save.txt");

	boost::archive::text_oarchive oa(ofs);

	oa << base_case_network;
	ofs.close();

}

void timing(std::vector<int> layers)
{
        G_Neural_Network * player = new G_Neural_Network(layers);
        int n = 0;
        for (auto i : layers) { n += i; } // sum of NN nodes.

        // Run gpu stuff on shared space
        int nBlocks = 1;    // GPU thread blocks to run
        int blockDim = n;   // threads per block, should be 256 for best performance
	for (int ii = 0; ii < 100; ++ii) {
        	auto start_gpu = std::chrono::high_resolution_clock::now();
		evaluate<<<nBlocks, blockDim>>>(player);    // evaluate on gpu
		hipDeviceSynchronize();    // wait to finish evaluation
        	auto end_gpu = std::chrono::high_resolution_clock::now();
        	cout << std::chrono::duration<double, std::nano> (end_gpu - start_gpu).count() << endl;
	}


}
